#include <cstdio>
#include <cstdlib>
#include "hip/hip_runtime.h"

#include "kernels.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("CUDA failure %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(1);                                        \
  }                                                 \
} while(0)



int main(int argc, const char *argv[])
{
    // CUDACHECK(fp32_hammer(hipStreamDefault, 108));
    // CUDACHECK(l2_ld_hammer(hipStreamDefault, 108));
    // CUDACHECK(gmem_ld_hammer(hipStreamDefault, 108));
    // CUDACHECK(fp_hammer(hipStreamDefault, 108));
    // CUDACHECK(tensor_hammer(hipStreamDefault, 108));
    CUDACHECK(gmem_fp_hammer(hipStreamDefault, 108));
    CUDACHECK(hipDeviceSynchronize());

    hipDeviceReset();
    return 0;
}
